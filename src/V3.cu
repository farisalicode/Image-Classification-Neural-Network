#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define INPUT_SIZE 784
#define HIDDEN_SIZE 128
#define OUTPUT_SIZE 10
#define LEARNING_RATE 0.01
#define EPOCHS 3
#define BATCH_SIZE 64
#define NUM_CLASSES 10
#define THREADCOUNT 128  
//cuz 128 are the max nodes working parallel

// Timer function (CPU-based)
double get_time(clock_t start) {
    return (double)(clock() - start) / CLOCKS_PER_SEC;
}

// Allocate and free matrix (host-side)
double** allocateMatrix(int rows, int cols) {
    double** mat = (double**)malloc(rows * sizeof(double*));
    for (int i = 0; i < rows; i++) {
        mat[i] = (double*)malloc(cols * sizeof(double));
    }
    return mat;
}

void freeMatrix(double** mat, int rows) {
    for (int i = 0; i < rows; i++) free(mat[i]);
    free(mat);
}


// CUDA error checking macro
#define CUDA_CHECK(err) do { \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
} while(0)








// CUDA kernels
__global__ void forwardHiddenKernel(double* W1, double* b1, double* input, double* hidden, int inputSize) {

   //no reuse in weights in a block(for an image) so no point of keeping them in shared memory 
   //we can use the input image, cuz it is reused by all 128 neurons

    //every block has an image 
    int sample = blockIdx.y; //for which block?
    int i = blockIdx.x * blockDim.x + threadIdx.x; //what thread is it?

    __shared__ double input_shared[INPUT_SIZE];
   
    for (int j = threadIdx.x; j < inputSize; j += blockDim.x) 
    {
        input_shared[j] = input[sample*inputSize + j];
    } //even out of bound threads are loading


    __syncthreads();



    if (i < HIDDEN_SIZE) {
        double sum = b1[i];
        for (int j = 0; j < inputSize; j++) {
            sum += W1[i * inputSize + j] * input_shared[j];
        }
        hidden[sample * HIDDEN_SIZE + i] = (sum > 0) ? sum : 0;
    }
}

__global__ void forwardOutputKernel(double* W2, double* b2, double* hidden, double* output, int hiddenSize) {

   //will use shared memory for hidden layer here 
   //in a block there is no repeated usage of weights so no need of those



    int sample = blockIdx.y;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ double hidden_shared[HIDDEN_SIZE];
    for (int j = threadIdx.x; j < hiddenSize; j += blockDim.x) {
        hidden_shared[j] = hidden[sample * hiddenSize + j];
    }
    __syncthreads();

    if (i < OUTPUT_SIZE) {
        double sum = b2[i];
        for (int j = 0; j < hiddenSize; j++) {
            sum += W2[i * hiddenSize + j] * hidden_shared[j];
        }
        output[sample * OUTPUT_SIZE + i] = sum;
    }
}













//10 threads against each image
// for d_output = output - target
__global__ void backwardOutputGradKernel(double* output, double* target, double* d_output) {
    int sample = blockIdx.x;
    int i = threadIdx.x;

    if (i < OUTPUT_SIZE) {
        d_output[sample * OUTPUT_SIZE + i] = output[sample * OUTPUT_SIZE + i] - target[sample * OUTPUT_SIZE + i];
    }
}





//number of kernels against every image will be ... hidden size i guess
__global__ void backwardHiddenGradKernel(double* W2, double* d_output, double* hidden, double* d_hidden, int hiddenSize) {
    

    //doutput getting reused 

    int sample = blockIdx.y; //each image
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ double d_output_shared[OUTPUT_SIZE];
   
    for (int j = threadIdx.x; j < OUTPUT_SIZE; j += blockDim.x) 
    {
        d_output_shared[j] = d_output[sample * OUTPUT_SIZE + j];
    }
    __syncthreads();

    if (i < HIDDEN_SIZE) {
        double sum = 0;
        for (int j = 0; j < OUTPUT_SIZE; j++) {
            sum += W2[j * hiddenSize + i] * d_output_shared[j];
        }
        d_hidden[sample * HIDDEN_SIZE + i] = sum * (hidden[sample * HIDDEN_SIZE + i] > 0);
    }
}


//128 threads for each output (10 blocks) thing so.... for each row and each column 
//basically for the whole W2 matrix
__global__ void updateW2Kernel(double* W2, double* b2, double* d_output, double* hidden, int hiddenSize) {

  //d_output and hidden getting reused in every block 
    __shared__ double d_output_shared[BATCH_SIZE];
    //all the threads in a block are reusing these values 

   
    int i = blockIdx.x;
    int j = threadIdx.x;

    if (j < BATCH_SIZE) {
        d_output_shared[j] = d_output[j * OUTPUT_SIZE + i];
    }
    
    __syncthreads(); // Make sure all threads finish loading before proceeding
    

    if (i < OUTPUT_SIZE && j < HIDDEN_SIZE) {
        double grad = 0;
        for (int s = 0; s < BATCH_SIZE; s++) {
            grad += d_output_shared[s] * hidden[s * hiddenSize + j];
        }
        W2[i * hiddenSize + j] -= LEARNING_RATE * grad / BATCH_SIZE;

        if (j == 0) {
            double b_grad = 0;
            for (int s = 0; s < BATCH_SIZE; s++) {
                b_grad += d_output_shared[s];
            }
            b2[i] -= LEARNING_RATE * b_grad / BATCH_SIZE;
        }
    }
}

__global__ void updateW1Kernel(double* W1, double* b1, double* d_hidden, double* input, int inputSize) {


   //now here similarly d_hidden is being reused 
   __shared__ double d_hidden_shared[BATCH_SIZE];

    int i = blockIdx.x;
    int j = threadIdx.x;

    if (j < BATCH_SIZE) {
        d_hidden_shared[j] = d_hidden[j * HIDDEN_SIZE + i];
    }

   __syncthreads();
 

    if (i < HIDDEN_SIZE && j < INPUT_SIZE) {
        double grad = 0;
        for (int s = 0; s < BATCH_SIZE; s++) {
            grad += d_hidden_shared[s] * input[s * inputSize + j];
        }
        W1[i * inputSize + j] -= LEARNING_RATE * grad / BATCH_SIZE;

        if (j == 0) {
            double b_grad = 0;
            for (int s = 0; s < BATCH_SIZE; s++) {
                b_grad += d_hidden_shared[s];
            }
            b1[i] -= LEARNING_RATE * b_grad / BATCH_SIZE;
        }
    }
}


//one thread for an ouput layer, cuz just ten values so, one thread is sufficient
__global__ void softmaxKernel(double* output, int batchSize, int size) {
    int sample = blockIdx.y;
    
    if (sample < batchSize) {
        
        double sum = 0.0;
        double temp = 0;
        for (int i = 0; i < size; i++) {
            temp = exp(output[sample * size + i]);
            output[sample * size + i] = temp;
            sum += temp;
        }

        for (int i = 0; i < size; i++) {
            output[sample * size + i] /= sum;
        }
    }
}




// Neural network structure
typedef struct {
    double *W1, *W2, *b1, *b2; // Device pointers
    double **W1_host, **W2_host; // Host pointers for initialization



} NeuralNetwork;

// Initialize network (host and device)
NeuralNetwork* createNetwork() {
    NeuralNetwork* net = (NeuralNetwork*)malloc(sizeof(NeuralNetwork));
    net->W1_host = allocateMatrix(HIDDEN_SIZE, INPUT_SIZE);
    net->W2_host = allocateMatrix(OUTPUT_SIZE, HIDDEN_SIZE);

    srand(time(NULL));
    for (int i = 0; i < HIDDEN_SIZE; i++)
        for (int j = 0; j < INPUT_SIZE; j++)
            net->W1_host[i][j] = ((double)rand() / RAND_MAX) * 0.01;
    for (int i = 0; i < OUTPUT_SIZE; i++)
        for (int j = 0; j < HIDDEN_SIZE; j++)
            net->W2_host[i][j] = ((double)rand() / RAND_MAX) * 0.01;

    // Allocate device memory
    CUDA_CHECK(hipMalloc(&net->W1, HIDDEN_SIZE * INPUT_SIZE * sizeof(double)));
    CUDA_CHECK(hipMalloc(&net->W2, OUTPUT_SIZE * HIDDEN_SIZE * sizeof(double)));
    CUDA_CHECK(hipMalloc(&net->b1, HIDDEN_SIZE * sizeof(double)));
    CUDA_CHECK(hipMalloc(&net->b2, OUTPUT_SIZE * sizeof(double)));

    // Copy weights to device
    for (int i = 0; i < HIDDEN_SIZE; i++)
        CUDA_CHECK(hipMemcpy(net->W1 + i * INPUT_SIZE, net->W1_host[i], INPUT_SIZE * sizeof(double), hipMemcpyHostToDevice));
    for (int i = 0; i < OUTPUT_SIZE; i++)
        CUDA_CHECK(hipMemcpy(net->W2 + i * HIDDEN_SIZE, net->W2_host[i], HIDDEN_SIZE * sizeof(double), hipMemcpyHostToDevice));

    // Zero-initialize biases
    CUDA_CHECK(hipMemset(net->b1, 0, HIDDEN_SIZE * sizeof(double)));
    CUDA_CHECK(hipMemset(net->b2, 0, OUTPUT_SIZE * sizeof(double)));

    return net;
}









void train(NeuralNetwork* net, double** images, double** labels, int numImages) {
    double *input_batch, *hidden_batch, *output_batch, *target_batch, *d_output, *d_hidden;

    CUDA_CHECK(hipMalloc(&input_batch, BATCH_SIZE * INPUT_SIZE * sizeof(double)));
    CUDA_CHECK(hipMalloc(&hidden_batch, BATCH_SIZE * HIDDEN_SIZE * sizeof(double)));
    CUDA_CHECK(hipMalloc(&output_batch, BATCH_SIZE * OUTPUT_SIZE * sizeof(double)));
    CUDA_CHECK(hipMalloc(&target_batch, BATCH_SIZE * OUTPUT_SIZE * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_output, BATCH_SIZE * OUTPUT_SIZE * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_hidden, BATCH_SIZE * HIDDEN_SIZE * sizeof(double)));

    clock_t total_start = clock();
    for (int epoch = 0; epoch < EPOCHS; epoch++) {
        clock_t epoch_start = clock();
        double loss = 0.0;
        int correct = 0;

        for (int i = 0; i < numImages; i += BATCH_SIZE) {
                                        //idhar change kia ha increment
             
            int actualBatchSize = (i + BATCH_SIZE > numImages) ? (numImages - i) : BATCH_SIZE;

            double host_input[BATCH_SIZE * INPUT_SIZE];
            double host_target[BATCH_SIZE * OUTPUT_SIZE];
            double host_output[BATCH_SIZE * OUTPUT_SIZE];

            for (int b = 0; b < actualBatchSize; b++) {
                       //running each iteration for every image

                for (int j = 0; j < INPUT_SIZE; j++)
                    host_input[b * INPUT_SIZE + j] = images[i + b][j];
                for (int j = 0; j < OUTPUT_SIZE; j++)
                    host_target[b * OUTPUT_SIZE + j] = labels[i + b][j];
            }


            //giving all the images
            CUDA_CHECK(hipMemcpy(input_batch, host_input, actualBatchSize * INPUT_SIZE * sizeof(double), hipMemcpyHostToDevice));
            //giving their answers here 
            CUDA_CHECK(hipMemcpy(target_batch, host_target, actualBatchSize * OUTPUT_SIZE * sizeof(double), hipMemcpyHostToDevice));
                             //formality, its actually 1 block for each image
            dim3 grid_hidden((HIDDEN_SIZE + THREADCOUNT-1) / THREADCOUNT, actualBatchSize);
            forwardHiddenKernel<<<grid_hidden, THREADCOUNT>>>(net->W1, net->b1, input_batch, hidden_batch, INPUT_SIZE);
            //relu inside the kernel
            
            CUDA_CHECK(hipDeviceSynchronize());

            //only 10 threads needed though but may help for shared memory
            dim3 grid_output((OUTPUT_SIZE + THREADCOUNT-1) / THREADCOUNT, actualBatchSize);
            forwardOutputKernel<<<grid_output, THREADCOUNT>>>(net->W2, net->b2, hidden_batch, output_batch, HIDDEN_SIZE);
            CUDA_CHECK(hipDeviceSynchronize());
            
            //fixed hamza's Softmax

            dim3 softmax_grid(1, actualBatchSize);
            softmaxKernel<<<softmax_grid, 1>>>(output_batch, actualBatchSize, OUTPUT_SIZE);

            CUDA_CHECK(hipMemcpy(host_output, output_batch, actualBatchSize * OUTPUT_SIZE * sizeof(double), hipMemcpyDeviceToHost));




            //for backward now 



            //here updating hiddend/outputlyare weights
            backwardOutputGradKernel<<<actualBatchSize, OUTPUT_SIZE>>>(output_batch, target_batch, d_output);
            CUDA_CHECK(hipDeviceSynchronize());



            dim3 grid_hidden_grad((HIDDEN_SIZE + THREADCOUNT-1) / THREADCOUNT, actualBatchSize);
            //try to keep minimum threacount greater than hidden size
            backwardHiddenGradKernel<<<grid_hidden_grad, HIDDEN_SIZE>>>(net->W2, d_output, hidden_batch, d_hidden, HIDDEN_SIZE);
            CUDA_CHECK(hipDeviceSynchronize());






                             //10 blocks, with 128 threads each 
            updateW2Kernel<<<OUTPUT_SIZE, HIDDEN_SIZE>>>(net->W2, net->b2, d_output, hidden_batch, HIDDEN_SIZE);


            updateW1Kernel<<<HIDDEN_SIZE, INPUT_SIZE>>>(net->W1, net->b1, d_hidden, input_batch, INPUT_SIZE);
            CUDA_CHECK(hipDeviceSynchronize());






            for (int b = 0; b < actualBatchSize; b++) {
                int pred = 0, actual = 0;
                for (int j = 0; j < OUTPUT_SIZE; j++) {
                    if (host_output[b * OUTPUT_SIZE + j] > host_output[b * OUTPUT_SIZE + pred])
                        pred = j;
                    if (labels[i + b][j] > labels[i + b][actual])
                        actual = j;
                    loss -= labels[i + b][j] * log(host_output[b * OUTPUT_SIZE + j] + 1e-8);
                }
                if (pred == actual) correct++;
            }
        }

        printf("Epoch %d - Loss: %.4f - Train Accuracy: %.2f%% - Time: %.3fs\n",
               epoch + 1, loss / numImages, (correct / (double)numImages) * 100, get_time(epoch_start));
    }
    printf("Total training time: %.3fs\n", get_time(total_start));
}

void evaluate(NeuralNetwork* net, double** images, double** labels, int numImages) {
    double *input_batch, *hidden_batch, *output_batch;

    CUDA_CHECK(hipMalloc(&input_batch, BATCH_SIZE * INPUT_SIZE * sizeof(double)));
    CUDA_CHECK(hipMalloc(&hidden_batch, BATCH_SIZE * HIDDEN_SIZE * sizeof(double)));
    CUDA_CHECK(hipMalloc(&output_batch, BATCH_SIZE * OUTPUT_SIZE * sizeof(double)));

    int correct = 0;
    clock_t eval_start = clock();

    for (int i = 0; i < numImages; i += BATCH_SIZE) {
        int actualBatchSize = (i + BATCH_SIZE > numImages) ? (numImages - i) : BATCH_SIZE;

        double host_input[BATCH_SIZE * INPUT_SIZE];
        double host_output[BATCH_SIZE * OUTPUT_SIZE];

        for (int b = 0; b < actualBatchSize; b++) {
            for (int j = 0; j < INPUT_SIZE; j++)
                host_input[b * INPUT_SIZE + j] = images[i + b][j];
        }

        CUDA_CHECK(hipMemcpy(input_batch, host_input, actualBatchSize * INPUT_SIZE * sizeof(double), hipMemcpyHostToDevice));

        dim3 grid_hidden((HIDDEN_SIZE + THREADCOUNT-1) / THREADCOUNT, actualBatchSize);
        forwardHiddenKernel<<<grid_hidden, THREADCOUNT>>>(net->W1, net->b1, input_batch, hidden_batch, INPUT_SIZE);
        CUDA_CHECK(hipDeviceSynchronize());

        dim3 grid_output((OUTPUT_SIZE + THREADCOUNT-1) / THREADCOUNT, actualBatchSize);
        forwardOutputKernel<<<grid_output, THREADCOUNT>>>(net->W2, net->b2, hidden_batch, output_batch, HIDDEN_SIZE);
        CUDA_CHECK(hipDeviceSynchronize());

        dim3 softmax_grid(1, actualBatchSize);
        softmaxKernel<<<softmax_grid, 1>>>(output_batch, actualBatchSize, OUTPUT_SIZE);


        CUDA_CHECK(hipMemcpy(host_output, output_batch, actualBatchSize * OUTPUT_SIZE * sizeof(double), hipMemcpyDeviceToHost));

        for (int b = 0; b < actualBatchSize; b++) {
            int pred = 0, actual = 0;
            for (int j = 0; j < OUTPUT_SIZE; j++) {
                if (host_output[b * OUTPUT_SIZE + j] > host_output[b * OUTPUT_SIZE + pred])
                    pred = j;
                if (labels[i + b][j] > labels[i + b][actual])
                    actual = j;
            }
            if (pred == actual) correct++;
        }
    }

    double accuracy = (double)correct / numImages * 100.0;
    printf("Evaluation Accuracy: %.2f%% - Time: %.3fs\n", accuracy, get_time(eval_start));

    hipFree(input_batch);
    hipFree(hidden_batch);
    hipFree(output_batch);
}

// MNIST loading and freeing unchanged
double** loadMNISTImages(const char* filename, int numImages) {
    FILE* file = fopen(filename, "rb");
    if (!file) { printf("Error opening %s\n", filename); exit(1); }
    fseek(file, 16, SEEK_SET);
    double** images = allocateMatrix(numImages, INPUT_SIZE);
    for (int i = 0; i < numImages; i++) {
        for (int j = 0; j < INPUT_SIZE; j++) {
            unsigned char pixel;
            if (fread(&pixel, sizeof(unsigned char), 1, file) != 1) {
                fprintf(stderr, "Error: Failed to read pixel\n");
                fclose(file);
                exit(EXIT_FAILURE);
            }
            images[i][j] = pixel / 255.0;
        }
    }
    fclose(file);
    return images;
}

double** loadMNISTLabels(const char* filename, int numLabels) {
    FILE* file = fopen(filename, "rb");
    if (!file) { printf("Error opening %s\n", filename); exit(1); }
    fseek(file, 8, SEEK_SET);
    double** labels = allocateMatrix(numLabels, OUTPUT_SIZE);
    for (int i = 0; i < numLabels; i++) {
        unsigned char label;
        if (fread(&label, sizeof(unsigned char), 1, file) != 1) {
            fprintf(stderr, "Error: Failed to read label\n");
            fclose(file);
            exit(EXIT_FAILURE);
        }
        for (int j = 0; j < OUTPUT_SIZE; j++) {
            labels[i][j] = (j == label) ? 1.0 : 0.0;
        }
    }
    fclose(file);
    return labels;
}

void freeNetwork(NeuralNetwork* net) {
    freeMatrix(net->W1_host, HIDDEN_SIZE);
    freeMatrix(net->W2_host, OUTPUT_SIZE);
    CUDA_CHECK(hipFree(net->W1));
    CUDA_CHECK(hipFree(net->W2));
    CUDA_CHECK(hipFree(net->b1));
    CUDA_CHECK(hipFree(net->b2));
    free(net);
}

// Main function
int main() {
    printf("MNIST Neural Network with CUDA\n\n");

    double** train_images = loadMNISTImages("../data/train-images.idx3-ubyte", 60000);
    double** train_labels = loadMNISTLabels("../data/train-labels.idx1-ubyte", 60000);
    double** test_images = loadMNISTImages("../data/t10k-images.idx3-ubyte", 10000);
    double** test_labels = loadMNISTLabels("../data/t10k-labels.idx1-ubyte", 10000);


    NeuralNetwork* net = createNetwork();
    train(net, train_images, train_labels, 60000);
    evaluate(net, test_images, test_labels, 10000);

    freeNetwork(net);
    return 0;
}
